﻿#include "hip/hip_runtime.h"
#include ""
#include "Test.cuh"

int main(int argc, char * argv[]) {
    setlocale(LC_ALL, "");
    ctest::PQTest();
    ctest::StackTest();
    ctest::PhotonTest();
}