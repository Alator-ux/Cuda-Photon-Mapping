#include "Test.cuh"

void ctest::check_errors() {
	auto cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "������ ������������� ����������: %s\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}
}