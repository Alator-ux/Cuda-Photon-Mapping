#include "hip/hip_runtime.h"
#include "ModelConstructInfo.cuh"
#include "CudaUtils.cuh"
#include <typeinfo>

int model_type_to_primitive_size(ModelType type){
    int size = -1;
    if ((int)type < 2) {
        size = (int)type + 3;
    }
    return size;
}

void initialize_cuda_data(size_t size_in_bytes, void* cpu_data, void** cuda_data) {
    checkCudaErrors(hipMalloc(cuda_data, size_in_bytes));
    checkCudaErrors(hipMemcpy(*cuda_data, cpu_data, size_in_bytes, hipMemcpyHostToDevice));
}

void initialize_cuda_texture(size_t size_in_bytes, bool is_vec3, void* cpu_data, void** cuda_data,
    hipTextureObject_t& texture) {
    initialize_cuda_data(size_in_bytes, cpu_data, cuda_data);

    hipResourceDesc resource_desc = {};
    resource_desc.resType = hipResourceTypeLinear;
    resource_desc.res.linear.devPtr = *cuda_data;
    resource_desc.res.linear.sizeInBytes = size_in_bytes;
    resource_desc.res.linear.desc = is_vec3 ? hipCreateChannelDesc<float4>() : hipCreateChannelDesc<float2>();

    hipTextureDesc texture_desc = {};
    texture_desc.addressMode[0] = hipAddressModeClamp;
    texture_desc.filterMode = hipFilterModePoint;
    texture_desc.readMode = hipReadModeElementType;
    texture_desc.normalizedCoords = 0;

    checkCudaErrors(hipCreateTextureObject(&texture, &resource_desc, &texture_desc, nullptr));
}

ModelConstructInfo* ModelConstructInfo::vector_to_device(const std::vector<ModelConstructInfo>& mcis) {
    ModelConstructInfo* device_mci;
    cpm::vec3* cuda_positions, *cuda_normals, *cpu_positions, *cpu_normals;
    cpm::vec2* cuda_texcoords, *cpu_texcoords;

    checkCudaErrors(hipMalloc((void**)&device_mci, mcis.size() * sizeof(ModelConstructInfo)));
    for (int i = 0; i < mcis.size(); i++) {
        auto mci = mcis[i];
    
        
        size_t size_in_bytes = mci.size * sizeof(cpm::vec3);
        /*initialize_cuda_texture(size_in_bytes, true, mci.positions, (void**)&cuda_positions, mci.positions_texture);
        initialize_cuda_texture(size_in_bytes, true, mci.normals, (void**)&cuda_normals, mci.normals_texture);*/
        initialize_cuda_data(size_in_bytes, mci.positions, (void**)&cuda_positions);
        initialize_cuda_data(size_in_bytes, mci.normals, (void**)&cuda_normals);

        size_in_bytes = mci.size * sizeof(cpm::vec2);
        //initialize_cuda_texture(size_in_bytes, false, mci.texcoords, (void**)&cuda_texcoords, mci.texcoords_texture);
        initialize_cuda_data(size_in_bytes, mci.texcoords, (void**)&cuda_texcoords);

        cpu_positions = mci.positions;
        cpu_texcoords = mci.texcoords;
        cpu_normals = mci.normals;
        mci.positions = cuda_positions;
        mci.texcoords = cuda_texcoords;
        mci.normals = cuda_normals;

        checkCudaErrors(hipMemcpy(device_mci + i, &mci, sizeof(ModelConstructInfo), hipMemcpyHostToDevice));

        mci.positions = cpu_positions;
        mci.texcoords = cpu_texcoords;
        mci.normals = cpu_normals;
    }
    
    return device_mci;
}