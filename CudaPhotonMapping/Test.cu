#include "Test.cuh"

void ctest::check_errors() {
	auto cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "������ ������������� ����������: %s\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}
}

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

void ctest::TestAll() {
	ctest::PQTest();
	ctest::StackTest();
	ctest::PhotonTest();
	ctest::TreeTest();
	ctest::PhotonMapInsertTest();
	ctest::PhotonMapGetClosestTest();
	ctest::RayTracingTest();
}
