#include "Test.cuh"

void ctest::check_errors() {
	auto cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "������ ������������� ����������: %s\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}
}

void ctest::TestAll() {
	ctest::PQTest();
	ctest::StackTest();
	ctest::PhotonTest();
	ctest::TreeTest();
	ctest::PhotonMapInsertTest();
	ctest::PhotonMapGetClosestTest();
	ctest::RayTracingTest();
}
